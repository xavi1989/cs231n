#include "hip/hip_runtime.h"
//Udacity HW 4
//Radix Sorting

#include "utils.h"
#include <thrust/host_vector.h>

/* Red Eye Removal
   ===============
   
   For this assignment we are implementing red eye removal.  This is
   accomplished by first creating a score for every pixel that tells us how
   likely it is to be a red eye pixel.  We have already done this for you - you
   are receiving the scores and need to sort them in ascending order so that we
   know which pixels to alter to remove the red eye.

   Note: ascending order == smallest to largest

   Each score is associated with a position, when you sort the scores, you must
   also move the positions accordingly.

   Implementing Parallel Radix Sort with CUDA
   ==========================================

   The basic idea is to construct a histogram on each pass of how many of each
   "digit" there are.   Then we scan this histogram so that we know where to put
   the output of each digit.  For example, the first 1 must come after all the
   0s so we have to know how many 0s there are to be able to start moving 1s
   into the correct position.

   1) Histogram of the number of occurrences of each digit
   2) Exclusive Prefix Sum of Histogram
   3) Determine relative offset of each digit
        For example [0 0 1 1 0 0 1]
                ->  [0 1 0 1 2 3 2]
   4) Combine the results of steps 2 & 3 to determine the final
      output location for each element and move it there

   LSB Radix sort is an out-of-place sort and you will need to ping-pong values
   between the input and output buffers we have provided.  Make sure the final
   sorted results end up in the output buffer!  Hint: You may need to do a copy
   at the end.

 */

__global__ 
void scan_kernel(unsigned *d_in, unsigned int *d_out, const size_t size) {
    int mid = threadIdx.x + blockDim.x * blockIdx.x;
    if(mid >= size)
        return;
    d_out[mid] = d_in[mid];
    __syncthreads();

    for(int s = 1; s <= size; s *= 2) {
          int spot = mid - s; 
         
          unsigned int val = 0;
          if(spot >= 0)
              val = d_out[spot];
          __syncthreads();
          if(spot >= 0)
              d_out[mid] += val;
          __syncthreads();

    }

    d_out[mid] = d_out[mid] - d_in[mid];
}

__global__
void exclusive_scan_kernel(unsigned *d_in, unsigned int *d_out, const size_t size) {
  // 1d of blockDim
  int index = threadIdx.x + blockDim.x * blockIdx.x;
  if(index >= size)
    return;

  d_out[index] = 1; //d_in[index];
  __syncthreads();

  size_t s = 0;
  for(s=1; s<size; s<<=1) {
    if((index + 1)%(2*s) == 0) {
      if(index - s >= 0) {
        d_out[index] = d_out[index] + d_out[index - s];
      }
    }

    __syncthreads();
  }

  // downsweep
  if(index == size-1) {
    d_out[index] = 0;
  }

  __syncthreads();

  for(s = s/2; s>0; s>>=1) {
    if((index + 1) % (2*s) == 0) {
      if(index - s >=0) {
        int tmp = d_out[index - s];
        d_out[index - s] = d_out[index];
        d_out[index] = d_out[index] + tmp;
      }
    }

    __syncthreads();
  }
}

__global__
void predicate_kernel(unsigned int *d_in,
                      unsigned int *d_out,
                      const size_t size, unsigned int mask, int *count) {
    int id = threadIdx.x + blockDim.x * blockIdx.x;

    if(id >= size)
        return;

    unsigned int bit = d_in[id] & mask;

    if(count) { // get 0
        d_out[id] = bit == 0 ? 1 : 0;
        if(bit == 0)
          atomicAdd(count, 1);
    } else {
        d_out[id] = bit == 1 ? 1 : 0;
    }
}

__global__
void scatter_kernel(unsigned int* const d_inputVals,
               unsigned int* const d_inputPos,
               unsigned int* const d_outputVals,
               unsigned int* const d_outputPos,
               const size_t size,
               unsigned int *scan_in,
               unsigned int *offset_in, int* offset) {
    int id = threadIdx.x + blockDim.x * blockIdx.x;

    if(id >= size)
        return;

    if(scan_in[id] == 1) {
        int addr = offset_in[id] + offset == NULL?0:(*offset);
        d_outputVals[addr] = d_inputVals[id];
        d_outputPos[addr] = d_inputPos[id];
    }
}

#include <stdio.h>

// gives you a good max size for n/d
int get_max_size(int n, int d) {
    return (int)ceil( (float)n/(float)d ) + 1;
}

void your_sort(unsigned int* const d_inputVals,
               unsigned int* const d_inputPos,
               unsigned int* const d_outputVals,
               unsigned int* const d_outputPos,
               const size_t numElems)
{ 
  //TODO
  //PUT YOUR SORT HERE
  unsigned int *in = (unsigned int *)calloc(1, numElems * sizeof(unsigned int));
  unsigned int *offset = (unsigned int *)calloc(1, numElems * sizeof(unsigned int));

  checkCudaErrors(hipMemcpy(in, d_inputVals,  sizeof(unsigned int) * numElems, hipMemcpyDeviceToHost));

  printf("numElems %d \n", (int)numElems);
  for(int i=0; i<50; i++) {
    //printf("%d \n", in[i]);
  }

  dim3 blockdim(1024);
  dim3 griddim(numElems / 1024 + 1);

  unsigned int *d_in, *d_inPos, *d_out, *d_outPos, *d_scan, *d_offset;
  checkCudaErrors(hipMalloc(&d_in,     sizeof(int) * numElems));
  checkCudaErrors(hipMalloc(&d_inPos,  sizeof(int) * numElems));

  checkCudaErrors(hipMalloc(&d_out,    sizeof(int) * numElems));
  checkCudaErrors(hipMemset(d_out, 0,  sizeof(int) * numElems));

  checkCudaErrors(hipMalloc(&d_outPos,   sizeof(int) * numElems));
  checkCudaErrors(hipMemset(d_outPos, 0, sizeof(int) * numElems));

  checkCudaErrors(hipMalloc(&d_scan,   sizeof(int) * numElems));
  checkCudaErrors(hipMemset(d_scan, 0, sizeof(int) * numElems));

  checkCudaErrors(hipMalloc(&d_offset, sizeof(int) * numElems));
  checkCudaErrors(hipMemset(d_offset, 0, sizeof(int) * numElems));

  checkCudaErrors(hipMemcpy(d_in,    d_inputVals, sizeof(int) * numElems, hipMemcpyDeviceToDevice));
  checkCudaErrors(hipMemcpy(d_inPos, d_inputPos,  sizeof(int) * numElems, hipMemcpyDeviceToDevice));

  int *d_count;
  int count;
  checkCudaErrors(hipMalloc(&d_count, sizeof(int)));

  unsigned int mask = 0x01;
  unsigned int *tmp;

  for(int j=0; j<32; j++) {
    // predicative kernel
    printf("j is %d 111111111111\n", j);
    mask <<= j;
    // bit 0
    count = 0;
    checkCudaErrors(hipMemcpy(d_count, &count,    sizeof(int), hipMemcpyHostToDevice));
    checkCudaErrors(hipGetLastError());
  checkCudaErrors(hipMemcpy(in, d_in,  sizeof(int) * numElems, hipMemcpyDeviceToHost));

  printf("numElems %d \n", (int)numElems);
  for(int i=0; i<10; i++) {
    //printf("%d \n", in[i]);
  }

    predicate_kernel<<<griddim, blockdim>>>(d_in, d_scan, numElems, mask , d_count);
    hipDeviceSynchronize();

  checkCudaErrors(hipMemcpy(offset, d_scan,  sizeof(unsigned int) * numElems, hipMemcpyDeviceToHost));

  printf("0 1 array \n");
  for(int i=0; i<10; i++) {
    printf("%d \n", offset[i]);
  }
    checkCudaErrors(hipGetLastError());


    for(int i = 0; i < get_max_size(numElems, blockdim.x); i++) {
        exclusive_scan_kernel<<<dim3(1), blockdim>>>(d_scan, d_offset, numElems);
        hipDeviceSynchronize(); checkCudaErrors(hipGetLastError());
    
    }



    hipDeviceSynchronize();
    checkCudaErrors(hipGetLastError());

  checkCudaErrors(hipMemcpy(offset, d_offset,  sizeof(unsigned int) * numElems, hipMemcpyDeviceToHost));

    printf("offset \n");
  for(int i=0; i<10; i++) {
    printf("%d \n", offset[i]);
  }
    checkCudaErrors(hipMemcpy(&count, d_count,    sizeof(int), hipMemcpyDeviceToHost));
    printf("count is %d \n", count);

    scatter_kernel<<<griddim, blockdim>>>(d_in, d_inPos,
                                          d_out, d_outPos,
                                          numElems,
                                          d_scan,
                                          d_offset, NULL);

    hipDeviceSynchronize();
    checkCudaErrors(hipGetLastError());
    // bit 1
    predicate_kernel<<<griddim, blockdim>>>(d_in, d_scan, numElems, mask , NULL);
    hipDeviceSynchronize();
    checkCudaErrors(hipGetLastError());

    for(int i = 0; i < get_max_size(numElems, blockdim.x); i++) {
        exclusive_scan_kernel<<<dim3(1), blockdim>>>(d_scan, d_offset, numElems);
        hipDeviceSynchronize(); checkCudaErrors(hipGetLastError());
    
    }

    hipDeviceSynchronize();
    checkCudaErrors(hipGetLastError());

    scatter_kernel<<<griddim, blockdim>>>(d_in, d_inPos,
                                          d_out, d_outPos,
                                          numElems,
                                          d_scan,
                                          d_offset, d_count);
    hipDeviceSynchronize();
    checkCudaErrors(hipGetLastError());

    tmp = d_in;
    d_in = d_out;
    d_out = tmp;

    tmp = d_inPos;
    d_inPos = d_outPos;
    d_outPos = tmp;
  }


  hipMemcpy(d_outputVals, d_out,    sizeof(unsigned int)*numElems, hipMemcpyDeviceToDevice);
  hipMemcpy(d_outputPos,  d_outPos, sizeof(unsigned int)*numElems, hipMemcpyDeviceToDevice);

  if(d_in) {
    checkCudaErrors(hipFree(d_in));
  }

  if(d_out) {
    checkCudaErrors(hipFree(d_out));
  }

  if(d_inPos) {
    checkCudaErrors(hipFree(d_inPos));
  }

  if(d_outPos) {
    checkCudaErrors(hipFree(d_outPos));
  }

  if(d_scan) {
    checkCudaErrors(hipFree(d_scan));
  }

  if(d_offset) {
    checkCudaErrors(hipFree(d_offset));
  }

  if(d_count) {
    checkCudaErrors(hipFree(d_count));
  }

}
